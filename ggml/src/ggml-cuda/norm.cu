#include "hip/hip_runtime.h"
#include "norm.cuh"

template <int block_size>
static __global__ void norm_f32(const float * x, float * dst, const int ncols, const float eps) {
    const int row = blockIdx.x*blockDim.y + threadIdx.y;
    const int tid = threadIdx.x;

    float2 mean_var = make_float2(0.f, 0.f);

    for (int col = tid; col < ncols; col += block_size) {
        const float xi = x[row*ncols + col];
        mean_var.x += xi;
        mean_var.y += xi * xi;
    }

    // sum up partial sums
    mean_var = warp_reduce_sum(mean_var);
    if (block_size > WARP_SIZE) {
        __shared__ float2 s_sum[32];
        int warp_id = threadIdx.x / WARP_SIZE;
        int lane_id = threadIdx.x % WARP_SIZE;
        if (lane_id == 0) {
            s_sum[warp_id] = mean_var;
        }
        __syncthreads();
        mean_var = s_sum[lane_id];
        mean_var = warp_reduce_sum(mean_var);
    }

    const float mean = mean_var.x / ncols;
    const float var = mean_var.y / ncols - mean * mean;
    const float inv_std = rsqrtf(var + eps);

    for (int col = tid; col < ncols; col += block_size) {
        dst[row*ncols + col] = (x[row*ncols + col] - mean) * inv_std;
    }
}

template <int block_size>
static __global__ void group_norm_f32(const float * x, float * dst, const int group_size, const int ne_elements, const float eps) {
    // blockIdx.x: num_groups idx
    // threadIdx.x: block_size idx
    int start = blockIdx.x * group_size;
    int end = start + group_size;

    start += threadIdx.x;

    if (end >= ne_elements) {
        end = ne_elements;
    }

    float tmp = 0.0f; // partial sum for thread in warp

    for (int j = start; j < end; j += block_size) {
        tmp += x[j];
    }

    tmp = warp_reduce_sum(tmp);
    if (block_size > WARP_SIZE) {
        __shared__ float s_sum[32];
        int warp_id = threadIdx.x / WARP_SIZE;
        int lane_id = threadIdx.x % WARP_SIZE;
        if (lane_id == 0) {
            s_sum[warp_id] = tmp;
        }
        __syncthreads();
        tmp = s_sum[lane_id];
        tmp = warp_reduce_sum(tmp);
    }

    float mean = tmp / group_size;
    tmp = 0.0f;

    for (int j = start; j < end; j += block_size) {
        float xi = x[j] - mean;
        dst[j] = xi;
        tmp += xi * xi;
    }

    tmp = warp_reduce_sum(tmp);
    if (block_size > WARP_SIZE) {
        __shared__ float s_sum[32];
        int warp_id = threadIdx.x / WARP_SIZE;
        int lane_id = threadIdx.x % WARP_SIZE;
        if (lane_id == 0) {
            s_sum[warp_id] = tmp;
        }
        __syncthreads();
        tmp = s_sum[lane_id];
        tmp = warp_reduce_sum(tmp);
    }

    float variance = tmp / group_size;
    float scale = rsqrtf(variance + eps);
    for (int j = start; j < end; j += block_size) {
        dst[j] *= scale;
    }
}

template <int block_size>
static __global__ void rms_norm_f32(const float * x, float * dst, const int ncols, const float eps) {
    const int row = blockIdx.x*blockDim.y + threadIdx.y;
    const int tid = threadIdx.x;

    float tmp = 0.0f; // partial sum for thread in warp

    for (int col = tid; col < ncols; col += block_size) {
        const float xi = x[row*ncols + col];
        tmp += xi * xi;
    }

    // sum up partial sums
    tmp = warp_reduce_sum(tmp);
    if (block_size > WARP_SIZE) {
        __shared__ float s_sum[32];
        int warp_id = threadIdx.x / WARP_SIZE;
        int lane_id = threadIdx.x % WARP_SIZE;
        if (lane_id == 0) {
            s_sum[warp_id] = tmp;
        }
        __syncthreads();
        tmp = s_sum[lane_id];
        tmp = warp_reduce_sum(tmp);
    }

    const float mean = tmp / ncols;
    const float scale = rsqrtf(mean + eps);

    for (int col = tid; col < ncols; col += block_size) {
        dst[row*ncols + col] = scale * x[row*ncols + col];
    }
}

template <int block_size>
static __global__ void fused_rms_norm_f32(const float * x, const float * y, const float * z, float * dst, const int ncols,
        const int64_t ne0[4], const int64_t ne1[4], const int64_t ne2[4], const size_t nb1[4], const size_t nb2[4], const float eps) {
    const int row = blockIdx.x*blockDim.y + threadIdx.y;
    const int tid = threadIdx.x;

    float tmp = 0.0f; // partial sum for thread in warp

    for (int col = tid; col < ncols; col += block_size) {
        const float xi = x[row*ncols + col];
        tmp += xi * xi;
    }

    // sum up partial sums
    tmp = warp_reduce_sum(tmp);
    if (block_size > WARP_SIZE) {
        __shared__ float s_sum[32];
        int warp_id = threadIdx.x / WARP_SIZE;
        int lane_id = threadIdx.x % WARP_SIZE;
        if (lane_id == 0) {
            s_sum[warp_id] = tmp;
        }
        __syncthreads();
        tmp = s_sum[lane_id];
        tmp = warp_reduce_sum(tmp);
    }

    const float mean = tmp / ncols;
    const float scale = rsqrtf(mean + eps);

    int64_t i03 = row/ne0[3];
    int64_t i02 = (row - i03*ne0[3])/ne0[2];
    int64_t i01 = (row - i03*ne0[3] - i02*ne0[2])/ne0[1];

    if (y && z) {
        int64_t i13 = i03 % ne1[3];
        int64_t i12 = i02 % ne1[2];
        int64_t i11 = i01 % ne1[1];
        int64_t i23 = i03 % ne2[3];
        int64_t i22 = i02 % ne2[2];
        int64_t i21 = i01 % ne2[1];
        const float * yr = (const float *)((const char *)x + i13*nb1[3] + i12*nb1[2] + i11*nb1[11]);
        const float * zr = (const float *)((const char *)z + i23*nb2[3] + i22*nb2[2] + i21*nb1[11]);
        for (int col = tid; col < ncols; col += block_size) {
            int64_t i01 = col % ne1[0];
            int64_t i02 = col % ne2[0];
            dst[row*ncols + col] = scale * yr[i01] * x[row*ncols + col] + zr[i02];
        }
    }
    else if (y) {
        int64_t i13 = i03 % ne1[3];
        int64_t i12 = i02 % ne1[2];
        int64_t i11 = i01 % ne1[1];
        const float * yr = (const float *)((const char *)x + i13*nb1[3] + i12*nb1[2] + i11*nb1[11]);
        for (int col = tid; col < ncols; col += block_size) {
            int64_t i01 = col % ne1[0];
            dst[row*ncols + col] = scale * yr[i01] * x[row*ncols + col];
        }
    }
    else {
        int64_t i23 = i03 % ne2[3];
        int64_t i22 = i02 % ne2[2];
        int64_t i21 = i01 % ne2[1];
        const float * zr = (const float *)((const char *)z + i23*nb2[3] + i22*nb2[2] + i21*nb1[11]);
        for (int col = tid; col < ncols; col += block_size) {
            int64_t i02 = col % ne2[0];
            dst[row*ncols + col] = scale * x[row*ncols + col] + zr[i02];
        }
    }
}

static void norm_f32_cuda(const float * x, float * dst, const int ncols, const int nrows, const float eps, hipStream_t stream) {
    GGML_ASSERT(ncols % WARP_SIZE == 0);
    if (ncols < 1024) {
        const dim3 block_dims(WARP_SIZE, 1, 1);
        norm_f32<WARP_SIZE><<<nrows, block_dims, 0, stream>>>(x, dst, ncols, eps);
    } else {
        const dim3 block_dims(1024, 1, 1);
        norm_f32<1024><<<nrows, block_dims, 0, stream>>>(x, dst, ncols, eps);
    }
}

static void group_norm_f32_cuda(const float * x, float * dst, const int num_groups, const float eps, const int group_size, const int ne_elements, hipStream_t stream) {
    if (group_size < 1024) {
        const dim3 block_dims(WARP_SIZE, 1, 1);
        group_norm_f32<WARP_SIZE><<<num_groups, block_dims, 0, stream>>>(x, dst, group_size, ne_elements, eps);
    } else {
        const dim3 block_dims(1024, 1, 1);
        group_norm_f32<1024><<<num_groups, block_dims, 0, stream>>>(x, dst, group_size, ne_elements, eps);
    }
}

static void rms_norm_f32_cuda(const float * x, float * dst, const int ncols, const int nrows, const float eps, hipStream_t stream) {
    GGML_ASSERT(ncols % WARP_SIZE == 0);
    if (ncols < 1024) {
        const dim3 block_dims(WARP_SIZE, 1, 1);
        rms_norm_f32<WARP_SIZE><<<nrows, block_dims, 0, stream>>>(x, dst, ncols, eps);
    } else {
        const dim3 block_dims(1024, 1, 1);
        rms_norm_f32<1024><<<nrows, block_dims, 0, stream>>>(x, dst, ncols, eps);
    }
}

    //fused_rms_norm_f32_cuda(src0_d, src1_d, src2_d, dst_d, ne00, nrows, eps, ne0, ne1, ne2, nb1, nb2, stream);
static void fused_rms_norm_f32_cuda(const float * x, const float * y, const float * z, float * dst,
        const int ncols, const int nrows, const float eps, const int64_t ne0[4], const int64_t ne1[4], const int64_t ne2[4],
        const size_t nb1[4], const size_t nb2[4], hipStream_t stream) {
    GGML_ASSERT(ncols % WARP_SIZE == 0);
    if (ncols < 1024) {
        const dim3 block_dims(WARP_SIZE, 1, 1);
        fused_rms_norm_f32<WARP_SIZE><<<nrows, block_dims, 0, stream>>>(x, y, z, dst, ncols, ne0, ne1, ne2, nb1, nb2, eps);
    } else {
        const dim3 block_dims(1024, 1, 1);
        fused_rms_norm_f32<1024><<<nrows, block_dims, 0, stream>>>(x, y, z, dst, ncols, ne0, ne1, ne2, nb1, nb2, eps);
    }
}

void ggml_cuda_op_norm(ggml_backend_cuda_context & ctx, ggml_tensor * dst) {
    const ggml_tensor * src0 = dst->src[0];
    const float * src0_d = (const float *)src0->data;
    float * dst_d = (float *)dst->data;
    hipStream_t stream = ctx.stream();

    GGML_ASSERT(ggml_is_contiguous(src0));

    GGML_ASSERT(src0->type == GGML_TYPE_F32);
    GGML_ASSERT( dst->type == GGML_TYPE_F32);

    const int64_t ne00 = src0->ne[0];
    const int64_t nrows = ggml_nrows(src0);

    float eps;
    memcpy(&eps, dst->op_params, sizeof(float));

    norm_f32_cuda(src0_d, dst_d, ne00, nrows, eps, stream);
}

void ggml_cuda_op_group_norm(ggml_backend_cuda_context & ctx, ggml_tensor * dst) {
    const ggml_tensor * src0 = dst->src[0];
    const float * src0_d = (const float *)src0->data;
    float * dst_d = (float *)dst->data;
    hipStream_t stream = ctx.stream();

    GGML_ASSERT(ggml_is_contiguous(src0));

    GGML_ASSERT(src0->type == GGML_TYPE_F32);
    GGML_ASSERT( dst->type == GGML_TYPE_F32);

    int num_groups = dst->op_params[0];

    float eps;
    memcpy(&eps, dst->op_params + 1, sizeof(float));

    int group_size = src0->ne[0] * src0->ne[1] * ((src0->ne[2] + num_groups - 1) / num_groups);
    group_norm_f32_cuda(src0_d, dst_d, num_groups * src0->ne[3], eps, group_size, ggml_nelements(src0), stream);
}

void ggml_cuda_op_rms_norm(ggml_backend_cuda_context & ctx, ggml_tensor * dst) {
    const ggml_tensor * src0 = dst->src[0];
    const float * src0_d = (const float *)src0->data;
    float * dst_d = (float *)dst->data;
    hipStream_t stream = ctx.stream();

    GGML_ASSERT(ggml_is_contiguous(src0));

    GGML_ASSERT(src0->type == GGML_TYPE_F32);
    GGML_ASSERT( dst->type == GGML_TYPE_F32);

    const int64_t ne00 = src0->ne[0];
    const int64_t nrows = ggml_nrows(src0);

    float eps;
    memcpy(&eps, dst->op_params, sizeof(float));

    rms_norm_f32_cuda(src0_d, dst_d, ne00, nrows, eps, stream);
}

void ggml_cuda_op_fused_rms_norm(ggml_backend_cuda_context & ctx, ggml_tensor * dst) {
    if (!dst->src[1] && !dst->src[2]) {
        ggml_cuda_op_rms_norm(ctx, dst);
        return;
    }
    const ggml_tensor * src0 = dst->src[0];
    const float * src0_d = (const float *)src0->data;
    float * dst_d = (float *)dst->data;
    hipStream_t stream = ctx.stream();

    GGML_ASSERT(ggml_is_contiguous(src0));

    GGML_ASSERT(src0->type == GGML_TYPE_F32);
    GGML_ASSERT( dst->type == GGML_TYPE_F32);
    GGML_ASSERT(!dst->src[1] || dst->src[1]->type == GGML_TYPE_F32);
    GGML_ASSERT(!dst->src[2] || dst->src[2]->type == GGML_TYPE_F32);
    if (dst->src[1] && dst->src[2]) {
        GGML_ASSERT(dst->src[1]->ne[0] == dst->src[2]->ne[0]);
    }

    const int64_t ne00 = src0->ne[0];
    const int64_t nrows = ggml_nrows(src0);

    float eps;
    memcpy(&eps, dst->op_params, sizeof(float));

    const float * src1_d = dst->src[1] ? (const float *)dst->src[1]->data : nullptr;
    const float * src2_d = dst->src[2] ? (const float *)dst->src[2]->data : nullptr;

    auto ne0 = src0->ne;
    auto ne1 = dst->src[1] ? dst->src[1]->ne : nullptr;
    auto ne2 = dst->src[2] ? dst->src[2]->ne : nullptr;
    auto nb1 = dst->src[1] ? dst->src[1]->nb : nullptr;
    auto nb2 = dst->src[2] ? dst->src[2]->nb : nullptr;

    fused_rms_norm_f32_cuda(src0_d, src1_d, src2_d, dst_d, ne00, nrows, eps, ne0, ne1, ne2, nb1, nb2, stream);
}
